#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdint>
#include <ostream>

#include "fbgemm_gpu/fbgemm_cuda_utils.cuh"
#include "fbgemm_gpu/permute_multi_embedding_function.h"

using Tensor = at::Tensor;

namespace fbgemm_gpu {

// Kernerl for permute pooled embedding op.
// This kernel is moving D elements per warp.
template <typename scalar_t>
__global__ void permute_multi_embs_kernel(
    const scalar_t** __restrict__ inputs,
    const scalar_t** __restrict__ outputs,
    const int64_t* __restrict__ permutes,
    const int64_t* __restrict__ input_lengths,
    const int64_t* __restrict__ output_lengths,
    const int64_t batch_size,
    const int64_t permute_size,
    const bool reverse_permute) {
  // workers in a warp handle a feature
  const int32_t worker_id = threadIdx.x % warpSize;
  const int32_t worker_size = warpSize;
  const int32_t permute_id =
      blockIdx.x * (blockDim.x / warpSize) + threadIdx.x / warpSize;
  const int32_t batch_id = blockIdx.y + gridDim.y * blockIdx.z;
  if (batch_id >= batch_size) {
    return;
  }
  if (permute_id >= permute_size) {
    return;
  }

  // parse permutes
  const int64_t params = 6;
  int64_t in_tensor, out_tensor, in_start, out_start, length, jump;
  if (reverse_permute) {
    out_tensor = permutes[params * permute_id];
    in_tensor = permutes[params * permute_id + 1];
    out_start = permutes[params * permute_id + 2];
    in_start = permutes[params * permute_id + 3];
  } else {
    in_tensor = permutes[params * permute_id];
    out_tensor = permutes[params * permute_id + 1];
    in_start = permutes[params * permute_id + 2];
    out_start = permutes[params * permute_id + 3];
  }
  length = permutes[params * permute_id + 4];
  jump = permutes[params * permute_id + 5];

  if (worker_id >= length) {
    return;
  }
  if (reverse_permute && jump < 0) {
    return;
  }

  // locate the batch_id
  int64_t in_length = input_lengths[in_tensor];
  scalar_t* input_ptr = (scalar_t*)inputs[in_tensor];
  input_ptr += batch_id * in_length;

  int64_t out_length = output_lengths[out_tensor];
  scalar_t* output_ptr = (scalar_t*)outputs[out_tensor];
  output_ptr += batch_id * out_length;

  // printf( // debug print
  //     "input_tensors[%ld][%ld][%d] = %f\n",
  //     in_tensor,
  //     batch_id,
  //     in_start + worker_id,
  //     input_ptr[in_start + worker_id]);
  if (fbgemm_gpu::is_aligned<fbgemm_gpu::Vec4T<scalar_t>>(
          &output_ptr[out_start]) &&
      fbgemm_gpu::is_aligned<fbgemm_gpu::Vec4T<scalar_t>>(
          &input_ptr[in_start])) {
    const int32_t vec_size = 4;
    const int32_t loop_end = length / (vec_size) * (vec_size);
    for (int32_t i = worker_id * vec_size; i < loop_end;
         i += worker_size * vec_size) {
      fbgemm_gpu::Vec4T<scalar_t>::copy(
          &input_ptr[in_start + i], &output_ptr[out_start + i]);
    }
    // Use elementwise access for the last incomplete vector.
    for (int32_t i = loop_end + worker_id; i < length; i += worker_size) {
      output_ptr[out_start + i] = input_ptr[in_start + i];
    }
  } else { // Fallback if not aligned.
    for (int32_t i = worker_id; i < length; i += worker_size) {
      output_ptr[out_start + i] = input_ptr[in_start + i];
    }
  }

  // for reverse_permute (backward) with jump
  while (reverse_permute && jump > 0 && jump < permute_size) {
    in_tensor = permutes[params * jump + 1];
    in_start = permutes[params * jump + 3];
    length = permutes[params * jump + 4];
    jump = -permutes[params * jump + 5];

    int64_t in_length = input_lengths[in_tensor];
    scalar_t* input_ptr = (scalar_t*)inputs[in_tensor];
    input_ptr += batch_id * in_length;

    for (int32_t i = worker_id; i < length; i += worker_size) {
      output_ptr[out_start + i] += input_ptr[in_start + i];
    }
  }
}

template <typename index_t>
Tensor from_vec(const std::vector<index_t> input) {
  const auto int_opts =
      torch::TensorOptions().dtype(torch::kInt64).pinned_memory(true);
  Tensor output = at::empty({static_cast<index_t>(input.size())}, int_opts);
  // Ensure that output is contiguous
  TORCH_CHECK(output.is_contiguous());
  std::memcpy(
      output.data_ptr<index_t>(), input.data(), input.size() * sizeof(index_t));
  return output;
}

template <typename scalar_t>
Tensor tensors_ptr(const at::TensorList& tensors) {
  auto size = tensors.size();
  Tensor ptr_tensor = at::empty(
      {static_cast<long>(size * sizeof(scalar_t*))},
      at::TensorOptions().dtype(tensors[0].scalar_type()).pinned_memory(true));

  // Ensure that ptr_tensor is contiguous
  TORCH_CHECK(ptr_tensor.is_contiguous());
  auto tp = reinterpret_cast<scalar_t**>(ptr_tensor.data_ptr());
  for (int32_t i = 0; i < tensors.size(); i++) {
    tp[i] = tensors[i].data_ptr<scalar_t>();
  }
  // Ensure that ptr_tensor is contiguous
  TORCH_CHECK(ptr_tensor.is_contiguous());
  return ptr_tensor;
}

std::vector<Tensor> permute_multi_embedding_gpu(
    const at::TensorList& pooled_embs,
    const std::vector<int64_t>& permutes,
    const std::vector<int64_t>& in_lengths,
    const std::vector<int64_t>& out_lengths,
    const bool& reverse_permute) {
  const int64_t permute_param = 6;
  int64_t num_of_input_tensors = in_lengths.size();
  int64_t num_of_output_tensors = out_lengths.size();
  int64_t batch_size = pooled_embs[0].size(0);
  int64_t permute_size = permutes.size() / permute_param;

  // check input tensors
  std::vector<Tensor> inputs;
  inputs.reserve(pooled_embs.size());
  for (int32_t i = 0; i < num_of_input_tensors; i++) {
    Tensor cont_tensor = pooled_embs[i].contiguous();
    inputs.push_back(cont_tensor);
    TENSORS_ON_SAME_DEVICE(cont_tensor, pooled_embs[i]);
    TENSORS_ON_SAME_DEVICE(pooled_embs[i], pooled_embs[0]);
  }

  // initiate output tensors
  std::vector<Tensor> outputs;
  outputs.reserve(num_of_output_tensors);
  for (int32_t i = 0; i < num_of_output_tensors; i++) {
    Tensor output =
        at::empty({batch_size, out_lengths[i]}, pooled_embs[0].options());
    outputs.push_back(output);
  }

  auto permutes_tensor = from_vec<int64_t>(permutes);
  auto in_lengths_tensor = from_vec<int64_t>(in_lengths);
  auto out_lengths_tensor = from_vec<int64_t>(out_lengths);

  auto device = pooled_embs[0].device();
  permutes_tensor = permutes_tensor.to(device, /*non_blocking=*/true);
  in_lengths_tensor = in_lengths_tensor.to(device, /*non_blocking=*/true);
  out_lengths_tensor = out_lengths_tensor.to(device, /*non_blocking=*/true);

  // This kernel is moving D elements per warp.
  // We are launching ( div_round_up(T, warp_per_block), B ) blocks.
  // The grid z dimension is also used by batch_size in case it's greater than
  // 65535.
  const int32_t warp_per_block =
      fbgemm_gpu::kMaxThreads / fbgemm_gpu::kWarpSize;
  const int32_t max_grid_dim_y =
      32768; // The CUDA maximum is 65535, not a power of 2.
  const dim3 threads(fbgemm_gpu::kMaxThreads);
  const dim3 blocks(
      fbgemm_gpu::div_round_up(permute_size, warp_per_block),
      std::min(static_cast<int32_t>(batch_size), max_grid_dim_y),
      (batch_size + max_grid_dim_y - 1) / max_grid_dim_y);

  FBGEMM_DISPATCH_FLOATING_TYPES(
      pooled_embs[0].scalar_type(), "permute_multi_embedding", [&] {
        Tensor in_tensor = tensors_ptr<scalar_t>(inputs);
        Tensor out_tensor = tensors_ptr<scalar_t>(outputs);
        in_tensor = in_tensor.to(device, /*non_blocking=*/true);
        out_tensor = out_tensor.to(device, /*non_blocking=*/true);
        permute_multi_embs_kernel<scalar_t>
            <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                (const scalar_t**)in_tensor.data_ptr(),
                (const scalar_t**)out_tensor.data_ptr(),
                permutes_tensor.data_ptr<int64_t>(),
                in_lengths_tensor.data_ptr<int64_t>(),
                out_lengths_tensor.data_ptr<int64_t>(),
                batch_size,
                permute_size,
                reverse_permute);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
  return outputs;
}

} // namespace fbgemm_gpu
